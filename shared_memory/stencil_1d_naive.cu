
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 1024
#define RADIUS 30
#define N 10000000 // size of 1D input array

__global__ void stencil_1d(int *in, int *out) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // detect out-of-bound 
    if (index >= N - 2 * RADIUS){
        return;
    }
    
    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++){
        result += in[index + RADIUS + offset]; 
    }

    // Store the result
    out[index] = result;
}

void random_ints(int *a, int n){
    int i;
    for (i = 0; i < n; ++i)
        //a[i] = rand();
        a[i] = 1;
}


int main(void) {
    int *h_in, *h_out;
    int *d_in, *d_out;
    int in_size = N * sizeof(int);
    int out_size = (N - 2 * RADIUS) * sizeof(int);

    h_in = (int *)malloc(in_size); random_ints(h_in, N);
    h_out = (int *)malloc(out_size);

    // initialize timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&d_in, in_size);
    hipMalloc((void **)&d_out, out_size);

    hipMemcpy(d_in, h_in, in_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, out_size, hipMemcpyHostToDevice);

    hipEventRecord(start);
    // Launch stencil_1d() kernel on GPU
    stencil_1d<<<(N - 2 * RADIUS + BLOCK_SIZE - 1) / BLOCK_SIZE,BLOCK_SIZE>>>(d_in, d_out);
    hipEventRecord(stop);

    // Copy result back to host
    hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost);

    // synchronization
    hipDeviceSynchronize();


    // print out the resulting array
    /*
    for (int i =0; i < N - 2 * RADIUS; i++) {
        printf("%d: %d", i, h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }
    */
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("used time: %fms\n", milliseconds);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
    return 0;
}
