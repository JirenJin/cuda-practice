
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 512
#define RADIUS 3
#define N 10 // size of 1D input array

__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x + blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // detect out-of-bound 
    //if (gindex >= N - RADIUS)
        //return;

    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in [gindex + BLOCK_SIZE];
    }
    
    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        //printf("%d", temp[lindex + offset]);
        result += temp[lindex + offset]; 

    // Store the result
    out[gindex] = result;
}

void random_ints(int *a, int n){
    int i;
    for (i = 0; i < n; ++i)
        //a[i] = rand();
        a[i] = 1;
}


int main(void) {
    int *h_in, *h_out;
    int *d_in, *d_out;
    int in_size = N * sizeof(int);
    int out_size = (N - 2 * RADIUS) * sizeof(int);

    h_in = (int *)malloc(in_size); random_ints(h_in, N);
    h_out = (int *)malloc(out_size);

    for (int i =0; i < N; i++) {
        printf("%d", h_in[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    hipMalloc((void **)&d_in, in_size);
    hipMalloc((void **)&d_out, out_size);

    hipMemcpy(d_in, h_in, in_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, out_size, hipMemcpyHostToDevice);

    // Launch stencil_1d() kernel on GPU
    stencil_1d<<<(N - 2 * RADIUS + BLOCK_SIZE - 1) / BLOCK_SIZE,BLOCK_SIZE>>>(d_in, d_out);

    // synchronization
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost);


    // print out the resulting array
    for (int i =0; i < N - 2 * RADIUS; i++) {
        printf("%d", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    // Clean up
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
    return 0;
}
